#include "hip/hip_runtime.h"
/*
 * GGUF Shard Atlas - CUDA Memory Management
 * 
 * Provides atomic memory swapping and atlas lookup for sharded GGUF files
 */

#include "gguf_shard_atlas.h"
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

// Atlas entry states
#define ATLAS_STATE_RESIDENT  0x01
#define ATLAS_STATE_SWAPPED   0x02
#define ATLAS_STATE_PENDING   0x04
#define ATLAS_STATE_LOCKED    0x08

// CUDA kernel for atomic page swapping
__global__ void kernel_atomic_swap_pages(
    atlas_entry_t* atlas,
    void* gpu_memory,
    void* swap_buffer,
    uint32_t* swap_queue,
    uint32_t queue_size,
    volatile uint32_t* completion_flag
) {
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t grid_size = gridDim.x * blockDim.x;
    
    // Cooperative group for synchronization
    auto block = cg::this_thread_block();
    auto grid = cg::this_grid();
    
    for (uint32_t i = tid; i < queue_size; i += grid_size) {
        uint32_t shard_id = swap_queue[i];
        atlas_entry_t* entry = &atlas[shard_id];
        
        // Atomic lock acquisition with backoff
        uint32_t expected = ATLAS_STATE_RESIDENT;
        uint32_t desired = ATLAS_STATE_RESIDENT | ATLAS_STATE_LOCKED;
        
        if (atomicCAS(&entry->state, expected, desired) == expected) {
            // Successfully acquired lock
            
            // Perform DMA copy with memory fence
            void* src = (char*)gpu_memory + entry->physical_addr;
            void* dst = (char*)swap_buffer + (shard_id * 4096);
            
            // Use cooperative copy for large pages
            if (blockDim.x >= 32) {
                cooperative_memcpy_async(block, dst, src, 4096);
            } else {
                memcpy(dst, src, 4096);
            }
            
            // Memory fence to ensure copy completion
            __threadfence();
            
            // Update atlas entry atomically
            entry->physical_addr = (uint64_t)dst;
            
            // Memory fence before state update
            __threadfence();
            
            // Release lock and update state
            atomicExch(&entry->state, ATLAS_STATE_SWAPPED);
            
            // Increment completion counter
            atomicAdd((uint32_t*)completion_flag, 1);
        }
    }
    
    // Grid-wide synchronization
    grid.sync();
}

// Host function for atlas lookup
extern "C" {

atlas_entry_t* atlas_lookup(atlas_t* atlas, uint64_t virtual_addr) {
    if (!atlas || !atlas->entries) {
        return nullptr;
    }
    
    // Binary search in sorted atlas
    uint32_t left = 0;
    uint32_t right = atlas->entry_count - 1;
    
    while (left <= right) {
        uint32_t mid = (left + right) / 2;
        atlas_entry_t* entry = &atlas->entries[mid];
        
        if (virtual_addr >= entry->virtual_addr && 
            virtual_addr < entry->virtual_addr + 4096) {
            return entry;
        }
        
        if (virtual_addr < entry->virtual_addr) {
            right = mid - 1;
        } else {
            left = mid + 1;
        }
    }
    
    return nullptr;
}

hipError_t atlas_atomic_swap(
    atlas_t* atlas,
    uint32_t* shard_ids,
    uint32_t count,
    hipStream_t stream
) {
    if (!atlas || !shard_ids || count == 0) {
        return hipErrorInvalidValue;
    }
    
    // Allocate device memory for swap queue
    uint32_t* d_swap_queue;
    hipError_t err = hipMalloc(&d_swap_queue, count * sizeof(uint32_t));
    if (err != hipSuccess) return err;
    
    // Copy swap queue to device
    err = hipMemcpyAsync(d_swap_queue, shard_ids, 
                          count * sizeof(uint32_t), 
                          hipMemcpyHostToDevice, stream);
    if (err != hipSuccess) {
        hipFree(d_swap_queue);
        return err;
    }
    
    // Allocate completion flag
    uint32_t* d_completion_flag;
    err = hipMalloc(&d_completion_flag, sizeof(uint32_t));
    if (err != hipSuccess) {
        hipFree(d_swap_queue);
        return err;
    }
    
    // Initialize completion flag
    uint32_t zero = 0;
    err = hipMemcpyAsync(d_completion_flag, &zero, sizeof(uint32_t),
                          hipMemcpyHostToDevice, stream);
    if (err != hipSuccess) {
        hipFree(d_swap_queue);
        hipFree(d_completion_flag);
        return err;
    }
    
    // Launch kernel with cooperative groups
    dim3 block_size(256);
    dim3 grid_size((count + block_size.x - 1) / block_size.x);
    
    // Ensure we don't exceed GPU limits
    int max_blocks_per_sm;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &max_blocks_per_sm, kernel_atomic_swap_pages, block_size.x, 0);
    
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, device);
    
    grid_size.x = min(grid_size.x, max_blocks_per_sm * props.multiProcessorCount);
    
    void* kernel_args[] = {
        &atlas->d_entries,
        &atlas->d_gpu_memory,
        &atlas->d_swap_buffer,
        &d_swap_queue,
        &count,
        &d_completion_flag
    };
    
    err = hipLaunchCooperativeKernel(
        (void*)kernel_atomic_swap_pages,
        grid_size, block_size,
        kernel_args, 0, stream
    );
    
    if (err != hipSuccess) {
        hipFree(d_swap_queue);
        hipFree(d_completion_flag);
        return err;
    }
    
    // Wait for completion
    err = hipStreamSynchronize(stream);
    
    // Cleanup
    hipFree(d_swap_queue);
    hipFree(d_completion_flag);
    
    return err;
}

hipError_t atlas_memory_fence(atlas_t* atlas) {
    // System-wide memory fence
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) return err;
    
    // Invalidate L2 cache for coherency
    err = hipDeviceSetCacheConfig(hipFuncCachePreferL1);
    if (err != hipSuccess) return err;
    
    // Memory barrier
    __sync_synchronize();
    
    return hipSuccess;
}

} // extern "C"

// Helper function for cooperative memory copy
__device__ void cooperative_memcpy_async(
    cg::thread_block block,
    void* dst, 
    const void* src, 
    size_t size
) {
    const size_t threads = block.size();
    const size_t tid = block.thread_rank();
    
    // Copy in 4-byte chunks using all threads
    const uint32_t* src32 = (const uint32_t*)src;
    uint32_t* dst32 = (uint32_t*)dst;
    const size_t words = size / 4;
    
    for (size_t i = tid; i < words; i += threads) {
        dst32[i] = src32[i];
    }
    
    // Handle remaining bytes
    if (tid == 0) {
        const size_t remaining = size % 4;
        const char* src_bytes = (const char*)src + (words * 4);
        char* dst_bytes = (char*)dst + (words * 4);
        
        for (size_t i = 0; i < remaining; i++) {
            dst_bytes[i] = src_bytes[i];
        }
    }
    
    // Synchronize block
    block.sync();
}
